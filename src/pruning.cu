#include "hip/hip_runtime.h"
/*  Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 *  Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 *  Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 *  of the code.
 */
#include "gpu.cuh"
#include "pruning.cuh"

namespace minkowski {

template <typename Dtype>
__device__ void device_memcpy(Dtype *dst, const Dtype *src, int num_elements) {
  for (int i = 0; i < num_elements; ++i)
    dst[i] = src[i];
}

template <typename Dtype, typename Itype>
__global__ void copy_in_out_map(const int n, const Dtype *in_feat,
                                Dtype *out_feat, const int nchannel,
                                const Itype *in_map, const Itype *out_map) {
  CUDA_KERNEL_LOOP(index, n) {
    device_memcpy(&out_feat[out_map[index] * nchannel],
                  &in_feat[in_map[index] * nchannel], nchannel);
  }
}

template <typename Dtype, typename Itype>
void PruningForwardKernelGPU(const Dtype *d_in_feat, Dtype *d_out_feat,
                             const int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
                             hipStream_t stream) {
  const int nnz = in_maps[0].size(0);

  copy_in_out_map<Dtype, Itype>
      <<<GET_BLOCKS(nnz), CUDA_NUM_THREADS, 0, stream>>>(
          nnz, d_in_feat, d_out_feat, nchannel, in_maps[0].data<Itype>(),
          out_maps[0].data<Itype>());
}

template <typename Dtype, typename Itype>
void PruningBackwardKernelGPU(Dtype *d_grad_in_feat,
                              const Dtype *d_grad_out_feat, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
                              hipStream_t stream) {
  const int nnz = in_maps[0].size(0);

  copy_in_out_map<Dtype, Itype>
      <<<GET_BLOCKS(nnz), CUDA_NUM_THREADS, 0, stream>>>(
          nnz, d_grad_out_feat, d_grad_in_feat, nchannel, out_maps[0].data<Itype>(),
          in_maps[0].data<Itype>());
}

template void PruningForwardKernelGPU<float, int32_t>(
    const float *d_in_feat, float *d_out_feat, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    hipStream_t stream);

template void PruningBackwardKernelGPU<float, int32_t>(
    float *d_grad_in_feat, const float *d_grad_out_feat, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    hipStream_t stream);

template void PruningForwardKernelGPU<double, int32_t>(
    const double *d_in_feat, double *d_out_feat, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    hipStream_t stream);

template void PruningBackwardKernelGPU<double, int32_t>(
    double *d_grad_in_feat, const double *d_grad_out_feat, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    hipStream_t stream);

} // end namespace minkowski
