#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA Corporation.
 * Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map.hpp"
#include "coordinate_map_cpu.hpp"
#include "coordinate_map_key.hpp"
#include "coordinate_map_manager.hpp"
#include "errors.hpp"
#include "types.hpp"
#include "utils.hpp"

#include "pooling_avg_kernel.cuh"
#include "pooling_max_kernel.cuh"

// Ninja
#include "local_pooling_cpu.cpp"

#include <pybind11/pybind11.h>
#include <torch/extension.h>

namespace minkowski {

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
std::pair<at::Tensor, at::Tensor> LocalPoolingForwardGPU(
    at::Tensor const &in_feat,
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    PoolingMode::Type pooling_mode,                    //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<coordinate_type, TemplatedAllocator> *p_map_manager) {

  ASSERT(in_feat.is_contiguous(), "in_feat must be contiguous");
  ASSERT(in_feat.is_cuda(), "in_feat must be on CUDA");
  ASSERT(in_feat.dim() == 2, "in_feat.dim():", in_feat.dim());

  coordinate_map_key_type in_key = p_in_map_key->get_key();
  ASSERT(p_map_manager->exists(in_key), ERROR_MAP_NOT_FOUND);

  ASSERT(in_feat.size(0) == p_map_manager->size(in_key), "Invalid in_feat size",
         in_feat.size(0), "!=", p_map_manager->size(in_key));

  // create an output coordinate map
  if (!p_out_map_key->is_key_set()) {
    coordinate_map_key_type out_key =
        std::get<0>(p_map_manager->stride(in_key, kernel_stride));
    p_out_map_key->set_key(out_key);
  }

  auto const &in_out = p_map_manager->kernel_map(
      p_in_map_key,    //
      p_out_map_key,   //
      kernel_size,     //
      kernel_stride,   //
      kernel_dilation, //
      region_type,     //
      offset, false /* is_transpose */, true /* is_pool */);

  auto const out_nrows = p_map_manager->size(p_out_map_key->get_key());
  at::Tensor out_feat =
      torch::zeros({out_nrows, in_feat.size(1)}, in_feat.options());
  LOG_DEBUG("Allocated", out_nrows, "x", in_feat.size(1), "features.");

  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  if (pooling_mode == PoolingMode::LOCAL_MAX_POOLING) {
    at::Tensor max_index = torch::empty({0}, torch::TensorOptions()
                                                 .device(in_feat.device())
                                                 .dtype(torch::kInt)
                                                 .requires_grad(false));
    max_index.resize_({out_nrows, in_feat.size(1)});
    max_index.zero_();
    TemplatedAllocator<char> byte_allocator;
    AT_DISPATCH_FLOATING_TYPES(
        in_feat.scalar_type(), "local_pooling_forward_gpu", [&] {
          MaxPoolingForwardKernelGPU<scalar_t, default_types::index_type,
                                     TemplatedAllocator<char>>(
              in_feat.template data_ptr<scalar_t>(),
              out_feat.template data_ptr<scalar_t>(), out_nrows,
              max_index.data_ptr<int>(), in_feat.size(1), in_out,
              byte_allocator, stream);
        });
    return std::make_pair(out_feat, max_index);

  } else {
    at::Tensor num_nonzero =
        torch::empty({0}, in_feat.options().requires_grad(false));

    if (pooling_mode == PoolingMode::LOCAL_AVG_POOLING) {
      num_nonzero.resize_({out_nrows});
      num_nonzero.zero_();
    }
    hipsparseHandle_t handle = getCurrentCUDASparseHandle();
    hipsparseSetStream(handle, stream);

    AT_DISPATCH_FLOATING_TYPES(
        in_feat.scalar_type(), "local_pooling_forward_gpu", [&] {
          TemplatedAllocator<char> byte_allocator;
          NonzeroAvgPoolingForwardKernelGPU<scalar_t, default_types::index_type,
                                            TemplatedAllocator<char>>(
              in_feat.template data_ptr<scalar_t>(), in_feat.size(0),
              out_feat.template data_ptr<scalar_t>(), out_nrows,
              num_nonzero.template data_ptr<scalar_t>(), in_feat.size(1),
              in_out, pooling_mode == PoolingMode::LOCAL_AVG_POOLING,
              byte_allocator, handle, stream);
        });

    return std::make_pair(out_feat, num_nonzero);
  }
}

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
at::Tensor LocalPoolingBackwardGPU(
    at::Tensor const &in_feat,                         //
    at::Tensor const &grad_out_feat,                   //
    at::Tensor const &num_nonzero,                     //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    PoolingMode::Type pooling_mode,                    //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<coordinate_type, TemplatedAllocator> *p_map_manager) {
  ASSERT(in_feat.is_contiguous(), "in_feat must be contiguous");
  ASSERT(grad_out_feat.is_contiguous(), "grad_out_feata must be contiguous");

  ASSERT(in_feat.is_cuda(), "in_feat must be on CUDA");
  ASSERT(grad_out_feat.is_cuda(), "in_feat must be on CUDA");

  ASSERT(in_feat.scalar_type() == grad_out_feat.scalar_type(), "type mismatch");

  ASSERT(in_feat.dim() == 2, "in_feat.dim():", in_feat.dim());
  ASSERT(grad_out_feat.dim() == 2, "grad_out_feat.dim():", grad_out_feat.dim());

  coordinate_map_key_type in_key = p_in_map_key->get_key();
  ASSERT(p_map_manager->exists(in_key), ERROR_MAP_NOT_FOUND);
  coordinate_map_key_type out_key = p_out_map_key->get_key();
  ASSERT(p_map_manager->exists(out_key), ERROR_MAP_NOT_FOUND);

  auto const &in_out = p_map_manager->kernel_map(
      p_in_map_key,    //
      p_out_map_key,   //
      kernel_size,     //
      kernel_stride,   //
      kernel_dilation, //
      region_type,     //
      offset, false /* is_transpose */, true /* is_pool */);

  at::Tensor grad_in_feat =
      torch::zeros({in_feat.size(0), in_feat.size(1)}, in_feat.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  if (pooling_mode == PoolingMode::LOCAL_MAX_POOLING) {
    AT_DISPATCH_FLOATING_TYPES(
        in_feat.scalar_type(), "local_pooling_backward_gpu", [&] {
          MaxPoolingBackwardKernelGPU<scalar_t>(
              grad_in_feat.template data_ptr<scalar_t>(), in_feat.size(0),
              grad_out_feat.template data_ptr<scalar_t>(),
              grad_out_feat.size(0), num_nonzero.data_ptr<int>(),
              in_feat.size(1), stream);
        });
  } else {
    AT_DISPATCH_FLOATING_TYPES(
        in_feat.scalar_type(), "local_pooling_backward_gpu", [&] {
          NonzeroAvgPoolingBackwardKernelGPU<
              scalar_t, default_types::index_type, TemplatedAllocator<char>>(
              grad_in_feat.template data_ptr<scalar_t>(), in_feat.size(0),
              grad_out_feat.template data_ptr<scalar_t>(),
              grad_out_feat.size(0), num_nonzero.template data_ptr<scalar_t>(),
              in_feat.size(1), in_out,
              pooling_mode == PoolingMode::LOCAL_AVG_POOLING, stream);
        });
  }

  return grad_in_feat;
}

// Forward
template std::pair<at::Tensor, at::Tensor>
LocalPoolingForwardGPU<default_types::dcoordinate_type,
                       detail::default_allocator>(
    at::Tensor const &in_feat,
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    PoolingMode::Type pooling_mode,                    //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::default_allocator>
        *p_map_manager);

template std::pair<at::Tensor, at::Tensor>
LocalPoolingForwardGPU<default_types::dcoordinate_type, detail::c10_allocator>(
    at::Tensor const &in_feat,
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    PoolingMode::Type pooling_mode,                    //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::c10_allocator>
        *p_map_manager);

// Backward
template at::Tensor LocalPoolingBackwardGPU<default_types::dcoordinate_type,
                                            detail::default_allocator>(
    at::Tensor const &in_feat,                         //
    at::Tensor const &grad_out_feat,                   //
    at::Tensor const &num_nonzero,                     //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    PoolingMode::Type pooling_mode,                    //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::default_allocator>
        *p_map_manager);

template at::Tensor
LocalPoolingBackwardGPU<default_types::dcoordinate_type, detail::c10_allocator>(
    at::Tensor const &in_feat,                         //
    at::Tensor const &grad_out_feat,                   //
    at::Tensor const &num_nonzero,                     //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    PoolingMode::Type pooling_mode,                    //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::c10_allocator>
        *p_map_manager);

} // end namespace minkowski
