#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA CORPORATION.
 * Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include <cstdio>
#include <iomanip>
#include <iostream>

#include "gpu.cuh"

namespace minkowski {

template <typename Dtype> void print(const thrust::device_vector<Dtype> &v) {
  for (size_t i = 0; i < v.size(); i++)
    std::cout << " " << std::fixed << std::setprecision(3) << v[i];
  std::cout << "\n";
}

template void print(const thrust::device_vector<float> &v);
template void print(const thrust::device_vector<int32_t> &v);

template <typename Dtype1, typename Dtype2>
void print(const thrust::device_vector<Dtype1> &v1,
           const thrust::device_vector<Dtype2> &v2) {
  for (size_t i = 0; i < v1.size(); i++)
    std::cout << " (" << v1[i] << "," << std::setw(2) << v2[i] << ")";
  std::cout << "\n";
}

template void print(const thrust::device_vector<int32_t> &v1,
                    const thrust::device_vector<int32_t> &v2);

const char *cublasGetErrorString(hipblasStatus_t error) {
  switch (error) {
  case HIPBLAS_STATUS_SUCCESS:
    return "HIPBLAS_STATUS_SUCCESS";
  case HIPBLAS_STATUS_NOT_INITIALIZED:
    return "HIPBLAS_STATUS_NOT_INITIALIZED";
  case HIPBLAS_STATUS_ALLOC_FAILED:
    return "HIPBLAS_STATUS_ALLOC_FAILED";
  case HIPBLAS_STATUS_INVALID_VALUE:
    return "HIPBLAS_STATUS_INVALID_VALUE";
  case HIPBLAS_STATUS_ARCH_MISMATCH:
    return "HIPBLAS_STATUS_ARCH_MISMATCH";
  case HIPBLAS_STATUS_MAPPING_ERROR:
    return "HIPBLAS_STATUS_MAPPING_ERROR";
  case HIPBLAS_STATUS_EXECUTION_FAILED:
    return "HIPBLAS_STATUS_EXECUTION_FAILED";
  case HIPBLAS_STATUS_INTERNAL_ERROR:
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
  case HIPBLAS_STATUS_NOT_SUPPORTED:
    return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
  case HIPBLAS_STATUS_UNKNOWN:
    return "HIPBLAS_STATUS_UNKNOWN";
#endif
  }
  return "Unknown cublas status";
}

const char *hipsparseGetErrorString(hipsparseStatus_t error) {
  // Read more at: http://docs.nvidia.com/cuda/cusparse/index.html#ixzz3f79JxRar
  switch (error) {
  case HIPSPARSE_STATUS_SUCCESS:
    return "The operation completed successfully.";
  case HIPSPARSE_STATUS_NOT_INITIALIZED:
    return "HIPSPARSE_STATUS_NOT_INITIALIZED";

  case HIPSPARSE_STATUS_ALLOC_FAILED:
    return "HIPSPARSE_STATUS_ALLOC_FAILED";

  case HIPSPARSE_STATUS_INVALID_VALUE:
    return "HIPSPARSE_STATUS_INVALID_VALUE";

  case HIPSPARSE_STATUS_ARCH_MISMATCH:
    return "HIPSPARSE_STATUS_ARCH_MISMATCH";

  case HIPSPARSE_STATUS_MAPPING_ERROR:
    return "HIPSPARSE_STATUS_MAPPING_ERROR";

  case HIPSPARSE_STATUS_EXECUTION_FAILED:
    return "HIPSPARSE_STATUS_EXECUTION_FAILED";

  case HIPSPARSE_STATUS_INTERNAL_ERROR:
    return "HIPSPARSE_STATUS_INTERNAL_ERROR";

  case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
    return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
  }

  return "<unknown>";
}

hipsparseHandle_t getCurrentCUDASparseHandle() {
  hipsparseHandle_t handle;
  CUSPARSE_CHECK(hipsparseCreate(&handle));
  return handle;
}

static std::string format_size(uint64_t size) {
  std::ostringstream os;
  os.precision(2);
  os << std::fixed;
  if (size <= 1024) {
    os << size << " bytes";
  } else if (size <= 1048576) {
    os << (size / 1024.0);
    os << " KiB";
  } else if (size <= 1073741824ULL) {
    os << size / 1048576.0;
    os << " MiB";
  } else {
    os << size / 1073741824.0;
    os << " GiB";
  }
  return os.str();
}

std::pair<size_t, size_t> get_memory_info() {
  size_t device_free;
  size_t device_total;
  CUDA_CHECK(hipMemGetInfo(&device_free, &device_total));
  return std::make_pair(device_total, device_free);
}

} // end namespace minkowski
