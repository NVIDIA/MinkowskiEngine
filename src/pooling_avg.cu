#include "hip/hip_runtime.h"
/* Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#ifndef GPU_POOLING_AVG
#define GPU_POOLING_AVG

#include <limits>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>

#include <thrust/functional.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include "gpu.cuh"
#include "pooling_avg.cuh"
#include "utils.hpp"

namespace minkowski {

template <typename Dtype>
__global__ void fill(const int n, Dtype *in_feat, Dtype val) {
  CUDA_KERNEL_LOOP(index, n) { in_feat[index] = val; }
}

template <typename Dtype>
__global__ void col2row_major(const int n, const int nrows, const int ncols,
                              const Dtype *colA, Dtype *rowA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index % nrows;
    j = index / nrows;
    rowA[i * ncols + j] = colA[index];
  }
}

template <typename Dtype>
__global__ void col2row_major_with_div(const int n, const int nrows,
                                       const int ncols,
                                       const Dtype *num_nonzero,
                                       const Dtype *colA, Dtype *rowA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index % nrows;
    j = index / nrows;
    if (num_nonzero[i]) {
      rowA[i * ncols + j] = colA[index] / num_nonzero[i];
    } else {
      rowA[i * ncols + j] = colA[index];
    }
  }
}

template <typename Dtype, typename Itype>
__global__ void set_gradient(const int n, const Dtype *d_grad_out,
                             Dtype *d_grad_in, const Itype *out_index,
                             int nchannel) {
  CUDA_KERNEL_LOOP(index, n) {
    atomicAdd(&d_grad_in[out_index[index]], d_grad_out[index]);
  }
}

template <typename Dtype, typename Itype>
__global__ void
set_gradient_nonzero(const int n, const Dtype *d_grad_out, Dtype *d_grad_in,
                     int nchannel, const Itype *in_map, const Itype *out_map) {
  CUDA_KERNEL_LOOP(index, n) {
    int nrow = index / nchannel;
    int ch = index % nchannel;
    atomicAdd(&d_grad_in[in_map[nrow] * nchannel + ch],
              d_grad_out[out_map[nrow] * nchannel + ch]);
  }
}

template <typename Dtype, typename Itype>
__global__ void
set_gradient_nonzero_avg(const int n, const Dtype *d_grad_out, Dtype *d_grad_in,
                         int nchannel, const Dtype *d_num_nonzero,
                         const Itype *in_map, const Itype *out_map) {
  CUDA_KERNEL_LOOP(index, n) {
    int nrow = index / nchannel;
    int ch = index % nchannel;
    int curr_num_nonzero = d_num_nonzero[out_map[nrow]];
    if (curr_num_nonzero > 0)
      atomicAdd(&d_grad_in[in_map[nrow] * nchannel + ch],
                d_grad_out[out_map[nrow] * nchannel + ch] / curr_num_nonzero);
  }
}

template <typename Dtype, typename Itype>
void NonzeroAvgPoolingForwardKernelGPU(const Dtype *d_in_feat, int in_nrows,
                                       Dtype *d_out_feat, int out_nrows,
                                       Dtype *d_num_nonzero, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
                                       bool use_avg, hipsparseHandle_t cushandle,
                                       hipStream_t stream) {
  int nmaps = 0;
  const Dtype alpha = 1;
  const Dtype beta = 0;
  hipsparseMatDescr_t descr = 0;
  Itype *d_scr, *d_in_map, *d_out_map, *d_csr_row;
  Dtype *d_ones, *d_csr_val, *d_tmp_out_feat;

  // Copy all maps to one vector
  for (const auto &map : in_maps)
    nmaps += map.size(0);

  /* Map prep */
  // Create d in map
  CUDA_CHECK(hipMalloc((void **)&d_scr,
                        (2 * nmaps + out_nrows + 1) * sizeof(Itype)));
  d_in_map = d_scr;  // n_maps
  d_out_map = d_scr + nmaps;     // n_maps
  d_csr_row = d_scr + 2 * nmaps; // out_nrows + 1

  CUDA_CHECK(hipMemcpy(d_in_map, in_maps[0].data<Itype>(), nmaps * sizeof(int),
                        hipMemcpyDeviceToDevice));

  CUDA_CHECK(hipMemcpy(d_out_map, out_maps[0].data<Itype>(), nmaps * sizeof(int),
                        hipMemcpyDeviceToDevice));

  /* sparse mm prep */
  CUDA_CHECK(hipMalloc((void **)&d_ones, ((use_avg ? in_nrows : 0) + nmaps +
                                           nchannel * out_nrows) *
                                              sizeof(Dtype)));

  if (use_avg) {
    // CUDA_CHECK(
    //     hipMalloc((void **)&d_ones,
    //                (in_nrows + nmaps + nchannel * out_nrows) *
    //                sizeof(Dtype)));
    d_ones = d_ones;                    // in_nrows;
    d_csr_val = d_ones + in_nrows;      // nmaps
    d_tmp_out_feat = d_csr_val + nmaps; // nchannel * out_nrows
    fill<Dtype><<<GET_BLOCKS(in_nrows), CUDA_NUM_THREADS, 0, stream>>>(
        in_nrows, d_ones, (Dtype)1.);
  } else {
    // CUDA_CHECK(hipMalloc((void **)&d_ones,
    //                       (nmaps + nchannel * out_nrows) * sizeof(Dtype)));
    d_csr_val = d_ones;                 // nmaps
    d_tmp_out_feat = d_csr_val + nmaps; // nchannel * out_nrows
  }

  fill<Dtype><<<GET_BLOCKS(nmaps), CUDA_NUM_THREADS, 0, stream>>>(
      nmaps, d_csr_val, (Dtype)1.);

  CUSPARSE_CHECK(hipsparseCreateMatDescr(&descr));
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  // Sort COO first
  sort_coo_gpu(cushandle, out_nrows, in_nrows, nmaps, d_out_map, d_in_map);

  // For CRS, sort row and col inds by row major.
  CUSPARSE_CHECK(hipsparseXcoo2csr(cushandle, d_out_map, nmaps, out_nrows,
                                  d_csr_row, HIPSPARSE_INDEX_BASE_ZERO));

  CUSPARSE_CHECK(
      cusparse_csrmm<Dtype>(cushandle,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(A)
                            HIPSPARSE_OPERATION_TRANSPOSE,     // op(B)
                            out_nrows,                        // M
                            nchannel,                         // N
                            in_nrows,                         // K
                            nmaps, &alpha, descr,
                            d_csr_val, // val
                            d_csr_row, // row
                            d_in_map,  // col
                            d_in_feat, // B
                            nchannel,  // ldb
                            &beta,
                            d_tmp_out_feat, // C
                            out_nrows       // ldc
                            ));

  if (use_avg) {
    CUSPARSE_CHECK(
        cusparse_csrmv<Dtype>(cushandle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, // op(A)
                              out_nrows,                        // M
                              in_nrows,                         // K
                              nmaps, &alpha, descr,
                              d_csr_val, // val
                              d_csr_row, // row
                              d_in_map,  // col
                              d_ones,    // B (in_nrows > out_nrows)
                              &beta,
                              d_num_nonzero)); // C

    col2row_major_with_div<Dtype>
        <<<GET_BLOCKS(out_nrows * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            out_nrows * nchannel, out_nrows, nchannel, d_num_nonzero,
            d_tmp_out_feat, d_out_feat);
  } else {
    col2row_major<Dtype>
        <<<GET_BLOCKS(out_nrows * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            out_nrows * nchannel, out_nrows, nchannel, d_tmp_out_feat,
            d_out_feat);
  }

  CUSPARSE_CHECK(hipsparseDestroyMatDescr(descr));

  hipFree(d_scr);
  hipFree(d_ones);

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}

template void NonzeroAvgPoolingForwardKernelGPU<float, int32_t>(
    const float *d_in_feat, int in_nrows, float *d_out_feat, int out_nrows,
    float *d_num_nonzero, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    bool use_avg,
    hipsparseHandle_t cushandle, hipStream_t stream);

template void NonzeroAvgPoolingForwardKernelGPU<double, int32_t>(
    const double *d_in_feat, int in_nrows, double *d_out_feat, int out_nrows,
    double *d_num_nonzero, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    bool use_avg,
    hipsparseHandle_t cushandle, hipStream_t stream);

template <typename Dtype, typename Itype>
void NonzeroAvgPoolingBackwardKernelGPU(
    Dtype *d_grad_in_feat, int in_nrows, const Dtype *d_grad_out_feat,
    int out_nrows, const Dtype *d_num_nonzero, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    bool use_avg, hipStream_t stream) {
  // d_grad_in_feat must be all set to 0

  int nmaps = 0;
  for (const auto &map : in_maps)
    nmaps += map.size(0);

  if (use_avg) {
    set_gradient_nonzero_avg<Dtype>
        <<<GET_BLOCKS(nmaps * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            nmaps * nchannel, d_grad_out_feat, d_grad_in_feat, nchannel,
            d_num_nonzero, in_maps[0].data<Itype>(), out_maps[0].data<Itype>());
  } else {
    set_gradient_nonzero<Dtype>
        <<<GET_BLOCKS(nmaps * nchannel), CUDA_NUM_THREADS, 0, stream>>>(
            nmaps * nchannel, d_grad_out_feat, d_grad_in_feat, nchannel,
            in_maps[0].data<Itype>(), out_maps[0].data<Itype>());
  }

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}

template void NonzeroAvgPoolingBackwardKernelGPU<float, int32_t>(
    float *d_grad_in_feat, int in_nrows, const float *d_grad_out_feat,
    int out_nrows, const float *d_num_nonzero, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    bool use_avg, hipStream_t stream);

template void NonzeroAvgPoolingBackwardKernelGPU<double, int32_t>(
    double *d_grad_in_feat, int in_nrows, const double *d_grad_out_feat,
    int out_nrows, const double *d_num_nonzero, int nchannel,
    const vector<at::Tensor>& in_maps, const vector<at::Tensor>& out_maps,
    bool use_avg, hipStream_t stream);

} // end namespace minkowski

#endif // end GPU_POOLING_AVG
